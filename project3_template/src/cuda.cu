#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 512;


int n_body;
int n_iteration;
std::chrono::duration<double> total_time;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
    if(x[i] <= radius2){
        vx[i] = -vx[i];
        x[i] = radius2 + err;
    }else if(x[i] >= bound_x - radius2){
        vx[i] = -vx[i];
        x[i] = bound_x - radius2 - err;
    }
    if(y[i] <= radius2){
        vy[i] = -vy[i];
        y[i] = radius2 + err;
    }else if(y[i] >= bound_y - radius2){
        vy[i] = -vy[i];
        y[i] = bound_y - radius2 - err;
    }
    x[i] += vx[i] * dt;
    y[i] += vy[i] * dt;
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) { 
        double ax = 0.0f;
        double ay = 0.0f;
        for(int j = 0; j < n; j++){
            // int signal = 1;
            if(index == j) continue;
            double dx = x[j] - x[i];
            double dy = y[j] - y[i];
            double d_square = dx * dx + dy * dy;
            
            //collision, v = -v, don't calculate force
            if(d_square <= 4*radius2*radius2) {
                d_square = 4*radius2 * radius2;

                // vx[i] = -vx[i];
                // vy[i] = -vy[i];
                // break;
            }
            // double d = sqrt(d_square);
            
            ax +=  gravity_const * m[j] * dx / (pow(d_square + err, 1.5));
            ay +=  gravity_const * m[j] * dy / (pow(d_square + err, 1.5));
        }
        vx[i] += ax * dt;
        vy[i] += ay * dt; 
    }
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        // x[i] = 2000.0f + rand() % (bound_x / 4);
        // y[i] = 2000.0f + rand() % (bound_y / 4);
        x[i] = rand() % bound_x;
        y[i] = rand() % bound_y;
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;
     
    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

<<<<<<< HEAD
        hipMemcpy(x, device_x, n_body*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body*sizeof(double), hipMemcpyDeviceToHost);

        
=======
        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);
>>>>>>> ec13d78b3723489de14cf9c8e9d2a9ffa8793bee

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        total_time += time_span;
        
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);
<<<<<<< HEAD
        l.save_frame(x, y);
=======

        l.save_frame(x, y);

>>>>>>> ec13d78b3723489de14cf9c8e9d2a9ffa8793bee
        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    master();

    printf("Student ID: 118010246\n"); // replace it with your student id
    printf("Name: Qin Lan\n"); // replace it with your name
    printf("Assignment 3: N Body Simulation CUDA Implementation\n");
    printf("Total time: %f; Average time: %f\n", total_time.count(), (total_time/(double)(n_iteration)).count());
    

    return 0;

}


